#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <random>
#include <fstream>
#include <stdio.h>

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif 

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

const double AU = 1.496e+11;
__device__ const double G = 6.67428e-11;
const double SCALE = 250 / AU;  // 1AU = 100 pixels
__device__ const double TIMESTEP = 3600 * 24 * 100; // 1 day


class Planet
{
    
public:
    double mass;
    double x;
    double y;
    double vx;
    double vy;
    Planet() : mass(1E+23), x(AU), y(AU) {}
    CUDA_CALLABLE_MEMBER Planet(double mass, double x, double y)
    {
        this->mass = mass;
        this->x = x;
        this->y = y;
        this->vx = 0;
        this->vy = 0;
    }
    CUDA_CALLABLE_MEMBER ~Planet()
    {
        //hipFree(mass);
    }

private:
};
class GravitySource
{

public:
    double mass;
    double x;
    double y;
    double vx;
    double vy;
    GravitySource() : mass(1E+30), x(0), y(0) {}
    CUDA_CALLABLE_MEMBER GravitySource(double mass, double x, double y)
    {
        this->mass = mass;
        this->x = x;
        this->y = y;
        this->vx = 0;
        this->vy = 0;
    }
    CUDA_CALLABLE_MEMBER ~GravitySource()
    {
    }

private:
};

__global__ void updatePositions(Planet* planets, int n, int t, double* x_coordinates, double* y_coordinates, GravitySource sun)
{
    // Indeks wątku
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Pamięć dzielona dla przechowywania pozycji planety
    __shared__ double x;
    __shared__ double y;

    // Jeśli indeks wątku jest mniejszy niż liczba planet
    if (i < n)
    {
        // Pobranie pozycji planety do pamięci dzielonej
        x = planets[i].x; 
        y = planets[i].y;
        // Synchronizacja wątków w bloku
        __syncthreads();
        

        // Obliczanie nowych pozycji dla planety
        //double F = 0;
        double Fx = 0;
        double Fy = 0;
        /*for (int j = 0; j < n; j++)
        {
            // Obliczanie siły działającej na planetę
            double r = distance(pos[0], pos[1], planets[j].x, planets[j].y);
            double F = G * planets[i].mass * planets[j].mass / (r * r);
            double theta = atan2(planets[j].y - pos[1], planets[j].x - pos[0]);
            Fx += F * cos(theta);
            Fy += F * sin(theta);
        }*/
        double distance_x = sun.x - planets[i].x;
        double distance_y = sun.y - planets[i].y;
        double r = sqrt(distance_x * distance_x + distance_y * distance_y);
        double theta = atan2(distance_y, distance_x);
        double F = G * sun.mass * planets[i].mass / (r * r);
        Fx = F * cos(theta);
        Fy = F * sin(theta); //z ta matma trzeba sprawdzic że sie wektor nie odwraca //to += chyb chociaz chuj wi
       
        // Aktualizacja pozycji planety
        planets[i].vx += Fx / planets[i].mass * TIMESTEP;
        planets[i].vy += Fy / planets[i].mass * TIMESTEP;
        planets[i].x += planets[i].vx * TIMESTEP;
        planets[i].y += planets[i].vy * TIMESTEP;
        //planets[i].x += 1e+12;
        //planets[i].y += 1e+11;

        // Zapis koordynatów do tablic
        //x_coordinates[t*n + i] = planets[i].x;
        //y_coordinates[t*n + i] = planets[i].y;
        x_coordinates[t * n + i] = planets[i].x;
        y_coordinates[t * n + i] = planets[i].y;
    }
}



int main()
{
    const int n = 100;
    const int units = 50; //jednostki czasowe
    const double MIN_X = -10.0 * AU;
    const double MAX_X = 10.0 * AU;
    const double MIN_Y = -10.0 * AU;
    const double MAX_Y = 10.0 * AU;
    const double MIN_MASS = 1E+23;
    const double MAX_MASS = 1E+25;
    std::uniform_real_distribution<> y_dis(MIN_Y, MAX_Y);
    std::uniform_real_distribution<> mass_dis(MIN_MASS, MAX_MASS);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> x_dis(MIN_X, MAX_X);
    Planet* planets = new Planet[n];;
    Planet* d_planets;
    GravitySource sun;
    sun = GravitySource(1E+30, 0, 0);
    //printf("masa = %.30f\n", G);
    //GravitySource* d_sun;
    //hipMalloc((void**)&d_sun, n);
    //planets = (Planet*)malloc(n);
    hipMalloc((void**)&d_planets, n);
    //Planet planets[n];
    for (int i = 0; i < n; i++)
    {
        double x = x_dis(gen);
        double y = y_dis(gen);
        double mass = mass_dis(gen);
        planets[i] = Planet(mass,x, y);
    }
    double* x_coordinates, * y_coordinates;
    double* d_x_coordinates, * d_y_coordinates;
    int size = (n * units) * sizeof(double);
    hipMalloc((void**)&d_x_coordinates, size);
    hipMalloc((void**)&d_y_coordinates, size);
    x_coordinates = (double*)malloc(size);
    y_coordinates = (double*)malloc(size);
    std::ofstream file("planets_coordinates3.csv");
    file << "x,y" << std::endl;
    for (int t = 0; t < units; t++) {

        hipMemcpy(d_planets, planets, n, hipMemcpyHostToDevice);
        hipMemcpy(d_x_coordinates, x_coordinates, size, hipMemcpyHostToDevice);
        hipMemcpy(d_y_coordinates, y_coordinates, size, hipMemcpyHostToDevice);
        updatePositions << <1000, 1000 >> > (d_planets, n, t, d_x_coordinates, d_y_coordinates, sun);
        hipMemcpy(planets, d_planets, n, hipMemcpyDeviceToHost);
        hipMemcpy(x_coordinates, d_x_coordinates, size, hipMemcpyDeviceToHost);
        hipMemcpy(y_coordinates, d_y_coordinates, size, hipMemcpyDeviceToHost);
        //hipFree(d_planets);
        hipDeviceSynchronize();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        printf("x = %.1f ", x_coordinates[t * n]);
        printf("y = %.1f\n", y_coordinates[t * n]);
        file << x_coordinates[t * n] << "," << y_coordinates[t * n] << std::endl;
        //printf("x = %.1f\n", x_coordinates[t * n + 1]);
        //printf("x = %.1f\n", x_coordinates[t * n + 2]);
        //printf("x = %.1f\n", x_coordinates[t * n + 3]);
        //printf("x = %.1f\n", x_coordinates[t * n + 4]);

    }
    
    //printf("x = %.1f\n", x_coordinates[2 * n + 1]);
    /*std::ofstream file("planets_coordinates2.csv");
    file << "x,y" << std::endl;
    for (int i = 0; i < n; i++)
    {
        file << planets[i].x << "," << planets[i].y << std::endl;
    }
    file.close();
    */
    printf("aaaaa");
    hipFree(d_planets);
    hipFree(d_x_coordinates);
    hipFree(d_y_coordinates);
    //free(planets);
    free(x_coordinates);
    free(y_coordinates);
    //for (int i = 0; i < n; i++)
    //{
    //    hipFree(d_planets + i);
    //    free(planets + i);
    //}
    delete[] planets;
    return 0;
}

